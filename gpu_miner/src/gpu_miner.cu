#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

#define MAX_NONCE 1e8
__constant__ BYTE d_difficulty_5_zeros[SHA256_HASH_SIZE] = "0000099999999999999999999999999999999999999999999999999999999999";

// TODO: Implement function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
struct Result {
    uint32_t nonce;
    BYTE hash[SHA256_HASH_SIZE * 2 + 1];
};


__device__ bool checkDifficulty(const BYTE* hash, const BYTE* difficulty) {
    for (int i = 0; i < SHA256_HASH_SIZE * 2; i++) {
        if (hash[i] < difficulty[i])
            return true;
        else if (hash[i] > difficulty[i])
            return false;
    }
    return true;
}

__device__ int volatile found = 0; // Global device variable to indicate if a nonce has been found

__global__ void findNonce(const BYTE* baseContent, size_t baseLength, Result* result, uint32_t maxNonce) {
    if (found) return;  // Check if a valid nonce has already been found
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    //printf("Testing with idx = %d\n", idx);

    BYTE buffer[BLOCK_SIZE];
    BYTE hash[SHA256_HASH_SIZE * 2 + 1];

    // Prepare the buffer with nonce
    memcpy(buffer, baseContent, baseLength);

    // Append the nonce to the end of the buffer
    char nonceStr[20];  // Ensure this is large enough to hold any 32-bit number
    int nonceLength = intToString(idx, nonceStr);
    memcpy(buffer + baseLength, nonceStr, nonceLength);

    // Calculate the total length of the buffer including the nonce
    size_t totalLength = baseLength + d_strlen((char*)buffer + baseLength);
    
    // Compute hash (single time hashing, n = 1)
    apply_sha256(buffer, totalLength, hash, 1);

    if (checkDifficulty(hash, d_difficulty_5_zeros)) {
        uint32_t old_nonce = atomicMin(&result->nonce, idx);
        if (result->nonce == idx) {
            memcpy(result->hash, hash, SHA256_HASH_SIZE * 2 + 1);
            atomicExch((int *)&found, 1);  // Set found to 1 to signal other threads
        }
    }
}


int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; // TODO: Update
	uint64_t nonce = 0; // TODO: Update
	size_t current_length;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	current_length = strlen((char*) block_content);


    Result h_result;
    Result *d_result;
    hipMalloc(&d_result, sizeof(Result));
    Result initial = {UINT32_MAX, {0}};    
    hipMemcpy(d_result, &initial, sizeof(Result), hipMemcpyHostToDevice);

    BYTE* d_block_content;
    hipMalloc((void**)&d_block_content, current_length * sizeof(BYTE));  // Allocate device memory for block_content

    // Copy data from host to device
    hipMemcpy(d_block_content, block_content, current_length * sizeof(BYTE), hipMemcpyHostToDevice);

    dim3 blockDim(256);
    dim3 gridDim((MAX_NONCE + blockDim.x - 1) / blockDim.x);

    hipEvent_t start, stop;
    startTiming(&start, &stop);

    findNonce<<<gridDim, blockDim>>>(d_block_content, current_length, d_result, MAX_NONCE); // Block content is precedent hash
    hipDeviceSynchronize();  // Ensure the kernel completes and data is ready, ensure all threads have completed their execution

    // Report timing and results
    float seconds = stopTiming(&start, &stop);

    // Copy the results back
    hipMemcpy(&h_result, d_result, sizeof(Result), hipMemcpyDeviceToHost);

    // Check the results
    if (h_result.nonce != UINT32_MAX) {
       printf("Nonce: %u, Hash: %s\n", h_result.nonce, h_result.hash);
    } else {
        printf("No valid nonce found.\n");
    }


    // Print results to file
    printResult(h_result.hash, h_result.nonce, seconds);
    hipFree(d_result);
    hipFree(d_block_content);


	return 0;
}
